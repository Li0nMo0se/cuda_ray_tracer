#include "hip/hip_runtime.h"
#include "rendering/engine.cuh"

namespace rendering
{

__device__ cuda_tools::Optional<space::IntersectionInfo>
Engine::cast_ray(const space::Ray& ray, const scene::Scene& scene)
{
    cuda_tools::Optional<space::IntersectionInfo> closest_intersection =
        cuda_tools::nullopt;

    // Find the closest intersection if it exists
    for (int32_t i = 0; i < scene.objects_.size_get(); i++)
    {
        const cuda_tools::Optional<space::IntersectionInfo> intersection =
            scene.objects_[i].intersect(ray);
        // If an intersection is found, assign this intersection if closer to
        // the origin of the ray
        if (intersection.has_value() &&
            (!closest_intersection || intersection.value().t_get() <
                                          closest_intersection.value().t_get()))
        {
            closest_intersection = intersection;
        }
    }

    // Compute the intersection point if there was an intersection
    if (closest_intersection.has_value())
        closest_intersection.value().compute_intersection(ray);
    return closest_intersection;
}

__device__ bool Engine::check_shadow(const scene::Scene& scene,
                                     const scene::Light& light,
                                     const space::Point3& intersection)
{
    const space::Vector3 vector_to_light = light.origin_get() - intersection;
    const space::Ray ray(intersection, vector_to_light.normalized());

    const cuda_tools::Optional<space::IntersectionInfo> intersection_info =
        cast_ray(ray, scene);

    if (!intersection_info.has_value())
        return false;

    // Is the intersection of the ray between the intersected point and the
    // light?
    // t_intersected is the distance between the intersected point and the
    // origin (that's the definition of a ray)
    const float distance_to_light = vector_to_light.length();
    return intersection_info.value().t_get() < distance_to_light;
}

__device__ color::Color3
Engine::get_object_color(const scene::Scene& scene,
                         const space::Ray& ray,
                         const space::IntersectionInfo& intersection_info)
{
    const scene::Object& obj = intersection_info.obj_get();
    const color::TextureMaterial& texture = obj.get_texture();
    const space::Vector3& intersection = intersection_info.intersection_get();

    const float kd = texture.get_kd(intersection);
    const float ks = texture.get_ks(intersection);
    const float ns = texture.get_ns(intersection);
    const color::Color3 obj_color = texture.get_color(intersection);

    // Normal of the object at the intersection point
    const space::Vector3& normal = obj.normal_get(ray, intersection_info);

    // Compute the reflected vector
    const space::Vector3 S =
        intersection - normal * 2 * intersection.dot(normal);

    color::Color3 color = color::black();

    for (int32_t i = 0; i < scene.lights_.size_get(); i++)
    {
        const scene::Light& light = scene.lights_[i];

        // Compute shadow (+ normal to avoid intersecting with yourself)
        if (check_shadow(scene, light, intersection))
            continue;

        const space::Vector3 L = light.origin_get() - intersection;
        const float intensity = light.intensity_get();
        // Compute the diffuse light
        const float coeff_diffuse =
            kd * normal.dot(L) * intensity * distance_attenuation(L.length());
        color += obj_color * coeff_diffuse;

        // Compute the specular light
        const float coeff_specular = ks * intensity * powf(S.dot(L), ns);
        if (coeff_specular > 0)
            color += coeff_specular;
    }

    return color;
}

__device__ inline float Engine::distance_attenuation(const float distance)
{
    return 1.f / distance;
}

__device__ color::Color3 Engine::cast_ray_color(const space::Ray& ray,
                                                const scene::Scene& scene)
{
    cuda_tools::Optional<space::IntersectionInfo> intersection =
        cast_ray(ray, scene);

    if (intersection.has_value())
    {
        space::IntersectionInfo& intersection_v = intersection.value();
        const scene::Object& intersected_obj = intersection_v.obj_get();
        // FIXME find more elegant way to do this
        intersection_v.auto_intersection_correction(
            intersected_obj.normal_get(ray, intersection_v));
        return get_object_color(scene, ray, intersection_v);
    }
    return color::background();
}

__device__ color::Color3
Engine::get_pixel_color(const space::Point3& curr_pixel,
                        const scene::Scene& scene,
                        const int32_t unit_x,
                        const int32_t unit_y,
                        const int32_t aliasing_level,
                        const int32_t reflection_max_depth)
{
    // FIXME: aliasing
    // FIXME: reflection

    const scene::Camera& camera = scene.camera_get();
    const space::Vector3 ray_direction =
        (curr_pixel - camera.origin_).normalized();
    const space::Ray ray(camera.origin_, ray_direction);

    return cast_ray_color(ray, scene);
}

struct FrameInfo
{
    const space::Point3 top_left;
    const float unit_x;
    const float unit_y;
};

// Copy arguments to have them in gpu registers, cache L1...
__global__ void kernel_render(DeviceImage<color::Color3> d_img,
                              const scene::Scene scene,
                              const FrameInfo frame_info,
                              const int32_t aliasing_level,
                              const int32_t reflection_max_depth)
{
    const int32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const int32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= d_img.width_get() || y >= d_img.height_get())
        return;

    const scene::Camera& camera = scene.camera_get();
    const space::Point3 curr_pixel = frame_info.top_left +
                                     x * (frame_info.unit_x * camera.x_axis_) -
                                     y * (frame_info.unit_y * camera.y_axis_);

    color::Color3* const data = d_img.data_get();
    // Ray computation with aliasing
    data[y * d_img.width_get() + x] =
        Engine::get_pixel_color(curr_pixel,
                                scene,
                                frame_info.unit_x,
                                frame_info.unit_y,
                                aliasing_level,
                                reflection_max_depth);
}

void Engine::render(const std::string& filename,
                    const int32_t resolution_width,
                    const int32_t resolution_height,
                    scene::Scene& scene,
                    const int32_t aliasing_level,
                    const int32_t reflection_max_depth)
{
    // Create Image
    ImageHandler<color::Color3> im(resolution_width, resolution_height);

    // Find width & height of a pixel
    const scene::Camera& camera = scene.camera_get();

    // Compute the height and width of the image in the 3D world
    const float height = std::tan(camera.alpha_) * camera.z_min_ * 2;
    const float width = std::tan(camera.beta_) * camera.z_min_ * 2;

    // Size of a pixel in the 3D world
    const float unit_x = width / resolution_width;
    const float unit_y = height / resolution_height;

    // Find top-left pixel
    // This space::Point3 will be used as a base for vector generation

    // P is the projection of `C` on the image plan
    const space::Point3 p = camera.origin_ + camera.z_axis_ * camera.z_min_;
    // Find the very top left point of the image in the 3D world
    space::Point3 top_left =
        p - (width / 2 * camera.x_axis_) + (height / 2 * camera.y_axis_);
    // Find the center of the top left pixel
    top_left = top_left + (unit_x / 2 * camera.x_axis_) -
               (unit_y / 2 * camera.y_axis_);

    // foreach pixel of the image
    //      Compute the ray from the origin of the camera to the pixel
    //      Find intersections of this ray with every objects of the scene
    //      (Calculate specular & diffuse contribution)
    constexpr int TILE_W = 32;
    constexpr int TILE_H = 8;
    constexpr dim3 block(TILE_W, TILE_H);
    const dim3 grid(1 + (resolution_width - 1) / block.x,
                    1 + (resolution_height - 1) / block.y);

    const FrameInfo frame_info{top_left, unit_x, unit_y};
    kernel_render<<<grid, block>>>(im.device,
                                   scene,
                                   frame_info,
                                   aliasing_level,
                                   reflection_max_depth);
    cuda_safe_call(hipDeviceSynchronize());
    check_error();

    // scene not usable because it has been copied

    // Retrive image
    im.save(filename);
}
} // namespace rendering