#include "hip/hip_runtime.h"
#include "sphere.cuh"
#include <cmath>

namespace scene
{
Sphere::Sphere(const space::Point3& origin,
               const float radius,
               const color::TextureMaterial* const texture)
    : Object(texture)
    , origin_(origin)
    , radius_(radius)
{
}

static __host__ __device__ cuda_tools::Optional<float>
solve_quadratic(const float a, const float b, const float c)
{
    float delta = b * b - 4 * a * c;
    if (delta < 0)
        return cuda_tools::nullopt;
    if (delta == 0)
        return (-b / (2 * a));

    // else delta > 0
    float t0 = (-b + sqrt(delta)) / (2 * a);
    float t1 = (-b - sqrt(delta)) / (2 * a);

    if (t0 < 0 && t1 < 0)
        return cuda_tools::nullopt;
    else if (t0 < 0)
        return t1;
    else if (t1 < 0)
        return t0;
    else // t1 and t0 positive
        return t0 < t1 ? t0 : t1;
}

cuda_tools::Optional<space::IntersectionInfo>
Sphere::intersect(const space::Ray& ray) const
{
    // P = O + tD
    // If sphere centered at (0, 0, 0)
    // ||P|| = R
    // R^2 = P^2
    // P^2 - R^2 = 0
    // (O + tD)^2 - R^2 = 0
    // O^2 + 2ODt + D^2t^2 - R^2 = 0
    // D^2t^2 + 2ODt + (O^2 - R^2) = 0

    // a = D.D == ||D||^2
    // b = 2* 0.D
    // c = O^2 - R^2

    // If sphere centered at C
    // a = D.D == ||D||^2
    // b = 2* (O - C).D
    // c = (O - C)^2 - R^2

    const space::Point3& ray_origin = ray.origin_get();
    const space::Vector3& ray_direction = ray.direction_get();

    const space::Vector3 L = ray_origin - this->origin_;
    const float a = ray_direction.dot(ray_direction);
    const float b = 2 * ray_direction.dot(L);
    const float c = L.dot(L) - radius_ * radius_;

    const cuda_tools::Optional<float> t_res = solve_quadratic(a, b, c);
    if (!t_res.has_value() || t_res.value() < space::T_MIN)
        return cuda_tools::nullopt;

    // Has intersected
    return space::IntersectionInfo(t_res.value(), *this);
}

space::Vector3
Sphere::normal_get(const space::Ray&,
                   const space::IntersectionInfo& intersection) const
{
    // p is the intersection point
    return (intersection.intersection_get() - origin_).normalized();
}

} // namespace scene