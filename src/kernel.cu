#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void print_kernel()
{
    printf("Hello from kernel !\n");
}

void print()
{
    print_kernel<<<1,1>>>();
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failure\n");
        fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(err));
    }
    else
        printf("Success\n");
}

